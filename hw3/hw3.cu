#include "hip/hip_runtime.h"
// =========================================================================================
// Name:  Chia Hsuan Wu
// SID:   42764118
// CSID:  y4d8
// =========================================================================================

#include <math.h>
#include <stdio.h>
#include "time_it.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>




/***************************************************************************************
*  print_vec: print the first few elements of a vector
*****************************************************************************************/

// CUDA_CALL and setup_kernel are from 
//   http://docs.nvidia.com/cuda/hiprand/device-api-overview.html#device-api-example
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__); \
  return EXIT_FAILURE;}} while(0)
  
void print_vec(float *x, uint n, const char *fmt, const char *who) {
  printf("%s = ", who);
  for(int i = 0; i < n; i++) {
    if(i > 0) printf(", ");
    printf(fmt, x[i]);
  }
  if(n > 10) printf(", ...");
  printf("\n");
}

void print_arr(uint *x, uint n, const char *fmt, const char *who) {
  printf("%s = ", who);
  for(int i = 0; i < n; i++) {
    if(i > 0) printf(", ");
    printf(fmt, x[i]);
  }
  if(n > 10) printf(", ...");
  printf("\n");
}




/***************************************************************************************
*  Question 1: GFlops
*****************************************************************************************/

// http://cuda-programming.blogspot.ca/2013/01/what-is-constant-memory-in-cuda.html
// declare constant memory
__constant__ float firstXvalue[1];

// arguments to the do_norm fuction as seen in perc.cu
struct logistic_arg {
	float *x;
	uint n;
	uint m;
};

// cuda version
__global__ void logisticKernel(float *x, int n, int m, float alpha) {
	int currentId = blockDim.x * blockIdx.x + threadIdx.x;
	if (currentId > 0 && currentId < n) {	
		// since the subsequent element will always be replaced by previous one
		// and the previous one was initally affected by x[0], then it is essentially saying
		// we do it offset*m times	

		float result = firstXvalue[0]; // cached, read-only access of global memory.
		
		int madM = currentId*m;

		// int madM = threadIdx.x*m;
		// at a different block, read from the previous block's last element
		// if (blockIdx.x > 0 && threadIdx.x == 0) {
		// 	result = x[blockDim.x * (blockIdx.x - 1) + (blockDim.x - 1)];
		// } 

		// do m*threadId times
		for (int i = 0; i < madM; ++i) {
			result = alpha*result*(1.0f - result);	// previous value
		}

		// write back to global
		x[currentId] = result;
		
	}
}

float* logistic(float *x, int n, int m) {
	float alpha = 3.6;
	float first[1];
	for (int i = 0; i < 1; ++i) {
		first[0] = x[i];
	}
	float *dev_x, *y;
	int size = n*sizeof(float);
	y = (float *)malloc(size);
	hipMalloc((void**)(&dev_x), size);
	hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(firstXvalue),first, sizeof(float));
	logisticKernel<<<ceil(n/256.0), 256>>>(dev_x, n, m, alpha); 
	hipMemcpy(y, dev_x, size, hipMemcpyDeviceToHost);
	hipFree(dev_x);
	return y;
}

void do_logistic(void *arg) {
	struct logistic_arg *logisticArg = (struct logistic_arg*)(arg);
	float* res = logistic(logisticArg->x, logisticArg->n, logisticArg->m);
	hipDeviceSynchronize();
}

// sequestial version
void logistic_ref(float *xf, int n, int m) {
	float alpha = 3.6;
	for (int i = 1; i < n; i++) {
		for (int j = 0; j < m; j++) {
			xf[i] = alpha*xf[i - 1]*(1.0f - xf[i - 1]);
		}
	}
	print_vec(xf, min(10, n), "%5.3f", "Logistic");
}





/***************************************************************************************
*  Question 2: GBps
*****************************************************************************************/

// arguments to the do_norm fuction as seen in perc.cu
struct norm_arg {
	float *x;
	uint n;
};

// multiply: read 1 write 1
__global__ void normKernal(float *x, uint n) {
	// store block data in here so we dont re read
	__shared__ float cache[256];

	uint dim = blockDim.x;
	uint blockBase = dim * blockIdx.x;
	uint tid = threadIdx.x ;
	uint myId = blockBase + tid;

	if(myId < n) {
		// x[myId] *= x[myId];	// write it directly back

		// store square version
		float val = x[myId];
		cache[tid] = val*val;

		// wait for everyone reach this point
		__syncthreads();	
		
		// if you are the first element of the block do this
		if (tid == 0) {
			float sum = 0.0;
			int adjusted = 256;
			// check if it is in the last block
			if (n - myId < 256) {
				adjusted = n - myId;
			}
			// sum up
			for (int i = 0; i < adjusted; i++) {
				sum += cache[i];
			}
			
			//write back
			x[blockIdx.x] = sum;
		}         
	}
}

// univ caller
float norm (float *x, uint n) {
	float *y;
	float *dev_x, *dev_y;
	uint blksize = 256;
	uint nblk = ceil(n/256.0);
	int size = n*sizeof(float);
	int Ysize = nblk*sizeof(float);
	y = (float *)malloc(size);
	hipMalloc((void**)(&dev_x), size);
	hipMalloc((void**)(&dev_y), size);
	hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);
	normKernal<<<nblk,blksize>>>(dev_x, n); 
	hipMemcpy(y, dev_x, Ysize, hipMemcpyDeviceToHost);

	// copy back and do the final sum up only nblock items
	float result = 0.0f;
	for (int i = 0; i < nblk; i++) {
		result += y[i];
	}
	result = sqrt(result);
	free(y);
	hipFree(dev_x);
	hipFree(dev_y);
	return result;
}

void do_norm(void *arg) {
	struct norm_arg *normArgument = (struct norm_arg*)(arg);
	float res = norm(normArgument->x, normArgument->n);
	hipDeviceSynchronize();
	// printf("Norm PAR = %f\n", res);
}

// seq version of the norm
void norm_ref (float *x, uint n) {
	float norm = 0.0;
	for (int i = 0; i < n; ++i) {
		norm += x[i]*x[i];
	}
	norm = sqrt(norm);
	printf("Norm = %f\n", norm);
}





/***************************************************************************************
*  Question 3
*****************************************************************************************/

// arguments to the rndm fuction as seen in perc.cu
struct rndm_arg {
  uint n,  // n = nblocks * threads per block * m elements 
       m;  // m elements
  hiprandState *dev_randState; // an array of n random number generators
  uint *dev_v; // write the final state here.
};

__global__ void setup_kernel(uint n, hiprandState *state) {
  uint myId = blockDim.x * blockIdx.x + threadIdx.x;

	// int myId = threadIdx.x + blockIdx.x * 256;	// offset count
  /* Each thread gets same seed, a different sequence 
     number, no offset */
  if(myId < n)
    hiprand_init(1234, myId, 0, &state[myId]);
}

// a modified version of 
// http://docs.nvidia.com/cuda/hiprand/device-api-overview.html#device-api-example
// with reference to perc.cu provided
__global__ void rndm(uint *v, uint n, hiprandState *state) {
	uint myId = blockDim.x*blockIdx.x + threadIdx.x;

	if (myId < n) {
		// Copy state to local memory
		hiprandState localState = state[myId];

		// Generate pseudo-random unsigned ints
		uint x = hiprand(&localState); 

		// Copy state back to global memory
		// this is becuase the number generate from the same state and the result will be the same
		// Since we want them to be different. We store the state back into global memory for
		// random number generations
		state[myId] = localState; 

		// Store results 
		v[myId] = x;
	}
}


void do_rndm(void *arg) {
	struct rndm_arg *my_arg = (struct rndm_arg*)(arg);
	rndm<<<1,256>>>(my_arg->dev_v, my_arg->n, my_arg->dev_randState);
	hipDeviceSynchronize();
}





/***************************************************************************************
*  MAIN
* ./hw3 1 n m 			 <- time logistics
* ./hw3 2 n any_number	 <- time norm
* ./hw3 3 m any_number   <- time random
*****************************************************************************************/

int main(int argc, char **argv) {
	uint choice = atoi(argv[1]);
	uint n = atoi(argv[2]);
	uint m = atoi(argv[3]);

	logistic_arg logParg;
	norm_arg parg;
	rndm_arg rdmParg;

	struct time_it_raw *tr = time_it_create(10);
  	struct time_it_stats stats;
  	float *x, *x_ref;
  	float result = 0.0;
  	int ops = 0;

  	int size = n*sizeof(float);
  	x = (float *)malloc(size);
	x_ref = (float *)malloc(size);

	for (int i = 0; i < n; i++) {
		x[i] = 0.123;
		x_ref[i] = 0.123;
	}

	switch (choice) {
		case 1:	// Question 1
			logParg.x = x;
			logParg.n = n;
			logParg.m = m;
			ops = 3*m*(n*(n - 1)/2);
			printf("\n");
			printf("Q1 Logistic\n");
			logistic_ref(x_ref, n, m);
			time_it_run(tr, do_logistic, (void *)(&logParg));
			time_it_get_stats(tr, &stats);
			result = ((ops/stats.mean)/pow(10,9)); // read + write
			printf("Logistic: n = %u, mean Time = %10.3le, std = %10.3le, GFps = %f\n", n, stats.mean, stats.std, result);
		break;
		case 2:
			printf("\n");
			printf("Q2 Norm\n");
			norm_ref(x_ref,n);
			parg.x = x;
			parg.n = n;
			time_it_run(tr, do_norm, (void *)(&parg));
			time_it_get_stats(tr, &stats);
			result = (((2*n*sizeof(float))/stats.mean)/pow(10,9));
			printf("Norm: n = %u, mean Time = %10.3le, std = %10.3le, GBps = %f\n", n, stats.mean, stats.std, result);
		break;
		case 3:
			m = 1;
			printf("\n");
			printf("Q3 Random\n");
			uint *v;
			m = n;
			n = 1 * 256 * n;
			if (n < 1000000) {
				n = 1000000;
				m = 1;
			}
			rdmParg.n = n;
			rdmParg.m = m;

			// allocate an array for the result on the CPU
			int vsz = (rdmParg.n)*sizeof(uint);
			v = (uint *)malloc(vsz);

			// allocate the result array and pseudo-random number generator states on the GPU
			CUDA_CALL(hipMalloc((void **)(&rdmParg.dev_v), vsz));
			CUDA_CALL(hipMalloc((void **)(&rdmParg.dev_randState), rdmParg.n*sizeof(hiprandState)));
			setup_kernel<<<1, 256>>>(rdmParg.n, rdmParg.dev_randState);

			// make the timing measurements.
			time_it_run(tr, do_rndm, (void *)(&rdmParg));

			// fetch the final state from the GPU
			hipMemcpy(v, rdmParg.dev_v, vsz, hipMemcpyDeviceToHost);
			time_it_get_stats(tr, &stats);
			result = (n/stats.mean);
			printf("rndm(%u, %u): pnps = %f, mean(T) = %10.3le, stddev(T) = %10.3le\n",
            	result, rdmParg.n, rdmParg.m, stats.mean, stats.std);
			print_arr(v, min(10, n), "%u", "RNDM V: ");
			CUDA_CALL(hipFree(rdmParg.dev_randState));
			CUDA_CALL(hipFree(rdmParg.dev_v));	
			free(v);
		break;
	}
	time_it_free(tr);
	free(x);
	free(x_ref);
	exit(0);
}